#include "devstreampool.h"
#include "cuda_helpers.h"
namespace cvl{
DevStreamPool::DevStreamPool(int size){ // 32 is the maximum number of async anyways...

    streams.resize(size*2);
    for(int i=0;i<size;++i)
        worked(hipStreamCreate(&streams[i])); // can crash but, if so with exit(1) error..., takes some time... dont allow copy? yeah
}
DevStreamPool::~DevStreamPool(){
    synchronize(); // might stall but more informative error if so than obscure device segfault...
   //for(int i=0;i<streams.size();++i)
     //   worked(hipStreamDestroy(streams[i]));
}
void DevStreamPool::synchronize(){
    for(int i=0;i<streams.size();++i)
        hipStreamSynchronize(streams[i]);
}
void DevStreamPool::synchronize(uint i){
    hipStreamSynchronize(streams[i % streams.size()]);
}
}
